#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdint.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <iostream>

hipError_t aes_on_cuda(const uint8_t *input, const uint8_t *keys, uint8_t *output, const size_t size, bool isEncrypt);

int hexToDec(char ch) {
	if (ch >= '0' && ch <= '9') {
		return ch - '0';
	}
	else if (ch >= 'a' && ch <= 'f') {
		return ch - 'a' + 10;
	}
	else if (ch >= 'A' && ch <= 'F') {
		return ch - 'A' + 10;
	}
	return -1;
}

inline size_t compute_block_size(size_t data_size) {
	return (data_size / 16 + 1) * 16;
}

size_t pkcs7_padding(uint8_t **data, size_t data_size) {
	// Compute the new text length
	size_t new_size = compute_block_size(data_size);
	// And recoad the diff between them.
	uint8_t diff = new_size - data_size;
	// Reallocate memory
	*data = (uint8_t *)realloc(*data, new_size);
	for (size_t index = data_size; index != new_size; ++index) {
		// And filling the extra uint8_ts with diff
		(*data)[index] = diff;
	}
	return new_size;
}

void encrypt_entry(std::string input_file, std::string output_file, uint8_t key[]) {
	// Open the file. if any errors were found, print error and quit.
	FILE *fp = fopen(input_file.c_str(), "rb+");
	if (fp == NULL) {
		printf("Error on opening file: %s.\n", input_file.c_str());
	}

	// Find out the length of file
	fseek(fp, 0, SEEK_END);
	size_t file_size = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// Declare two buffer pointer and allocate memory for plaintext
	uint8_t *data_in, *data_out;
	data_in = (uint8_t *)malloc(file_size * sizeof(uint8_t));

	// Read all file content to RAM.
	fread(data_in, file_size, 1, fp);

	// use PKCS#7 padding mode
	size_t full_block_size = pkcs7_padding(&data_in, file_size);

	// Allocate memory for chipertext, as original plaintext has been padded, the size is not the same.
	data_out = (uint8_t *)malloc(full_block_size * sizeof(uint8_t));

	// Record when it begin.
	clock_t begin = clock();

	// Copy memory and launch CUDA kernel
	hipError_t cudaStatus = aes_on_cuda(data_in, key, data_out, full_block_size, true);
	if (cudaStatus != hipSuccess) {
		printf("Something bad happened!\n");
	}


	// Record when it come to the end.
	clock_t end = clock();

	// And print the duration
	printf("Time cost: %ums\n", end - begin);

	// Finally, write chipertext to filesystem.
	FILE *out_fp = fopen(output_file.c_str(), "wb+");
	if (!out_fp) {
		// Can't open file.
		puts("fuck you");
	}
	else {
		fwrite(data_out, full_block_size, 1, out_fp);
	}

	// Release all using memory
	free(data_in);
	free(data_out);

	// Close all opened files.
	fclose(fp);
	fclose(out_fp);
}

void decrypt_entry(std::string input_file, std::string output_file, uint8_t key[]) {
	puts("Mode: Decrypt");
	// Open the file. if any errors were found, print error and quit.
	FILE *fp = fopen(input_file.c_str(), "rb+");
	if (fp == NULL) {
		printf("Error on opening file: %s.\n", input_file.c_str());
	}

	// Find out the length of file
	fseek(fp, 0, SEEK_END);
	size_t file_size = ftell(fp);
	fseek(fp, 0, SEEK_SET);

	// Declare two buffer pointer and allocate memory for plaintext
	uint8_t *data_in, *data_out;
	data_in = (uint8_t *)malloc(file_size * sizeof(uint8_t));

	// Read all file content to RAM.
	fread(data_in, file_size, 1, fp);

	// Allocate memory for chipertext, as original plaintext has been padded, the size is not the same.
	data_out = (uint8_t *)malloc(file_size * sizeof(uint8_t));

	// Record when it begin.
	clock_t begin = clock();

	// Copy memory and launch CUDA kernel
	hipError_t cudaStatus = aes_on_cuda(data_in, key, data_out, file_size, false);
	if (cudaStatus != hipSuccess) {
		printf("Something bad happened!\n");
	}

	// Record when it come to the end.
	clock_t end = clock();

	// And print the duration
	printf("Time cost: %ums\n", end - begin);

	// Find out padding value used by PKCS#7
	size_t diff = data_out[file_size - 1];

	// And excluding them from being part of our file
	size_t real_file_size = file_size - diff;

	// Finally, write chipertext to filesystem.
	FILE *out_fp = fopen(output_file.c_str(), "wb+");
	if (!out_fp) {
		// Can't open file.
		puts("fuck you");
	}
	else {
		fwrite(data_out, real_file_size, 1, out_fp);
	}

	// Release all using memory
	free(data_in);
	free(data_out);

	// Close all opened files.
	fclose(fp);
	fclose(out_fp);
}


int main(int argc, char *argv[])
{
	if (argc == 1) {
		// we need more parameters, print help message and quit.
		printf("useage: %s -m (encrypt|decrypt) -k 0123456789abcdeffedcba9876543210 -i /path/to/input -o /path/to/output -t <thread count>\n", strrchr(argv[0], '\\') + 1);
		return 1;
	}

	bool isEnctypt = false;
	std::string inputFile, outputFile;
	uint8_t key[16];
	int i, threadCount;
	for (i = 1; i < argc - 1; ++i) {
		if (std::string(argv[i]) == "-m") {
			i++;
			if (std::string(argv[i]) == "encrypt") {
				isEnctypt = true;
			}
			else if (std::string(argv[i]) == "decrypt") {
				isEnctypt = false;
			}
			else {
				printf("Mode must be encrypt or decrypt.");
				return 2;
			}
		}
		else if (std::string(argv[i]) == "-k") {
			// parse input key
			i++;
			int j;
			for (j = 0; j != 16; j += 2) {
				key[j / 2] = (hexToDec(argv[i][j]) << 8) & (hexToDec(argv[i][j]));
			}
		}
		else if (std::string(argv[i]) == "-i") {
			// parse input file
			i++;
			inputFile = std::string(argv[i]);
		}
		else if (std::string(argv[i]) == "-o") {
			// parse input file
			i++;
			outputFile = std::string(argv[i]);
		}
		else if (std::string(argv[i]) == "-t") {
			// parse thread count
			i++;
			sscanf(argv[i], "%d", &threadCount);
		}
	}

	if (isEnctypt) {
		encrypt_entry(inputFile, outputFile, key);
	}
	else {
		decrypt_entry(inputFile, outputFile, key);
	}

    hipDeviceReset();
	
    return 0;
}

// SBox and InverseSBox
#if 1

const uint8_t round_const[] = {
	0x01, 0x02, 0x04, 0x08, 0x10, 0x20, 0x40, 0x80, 0x1b, 0x36,
};

__device__  __host__ uint8_t sbox(uint8_t in) {
	const static uint8_t __sbox[] = {
		0x63, 0x7c, 0x77, 0x7b, 0xf2, 0x6b, 0x6f, 0xc5, 0x30, 0x01, 0x67, 0x2b, 0xfe, 0xd7, 0xab, 0x76,
		0xca, 0x82, 0xc9, 0x7d, 0xfa, 0x59, 0x47, 0xf0, 0xad, 0xd4, 0xa2, 0xaf, 0x9c, 0xa4, 0x72, 0xc0,
		0xb7, 0xfd, 0x93, 0x26, 0x36, 0x3f, 0xf7, 0xcc, 0x34, 0xa5, 0xe5, 0xf1, 0x71, 0xd8, 0x31, 0x15,
		0x04, 0xc7, 0x23, 0xc3, 0x18, 0x96, 0x05, 0x9a, 0x07, 0x12, 0x80, 0xe2, 0xeb, 0x27, 0xb2, 0x75,
		0x09, 0x83, 0x2c, 0x1a, 0x1b, 0x6e, 0x5a, 0xa0, 0x52, 0x3b, 0xd6, 0xb3, 0x29, 0xe3, 0x2f, 0x84,
		0x53, 0xd1, 0x00, 0xed, 0x20, 0xfc, 0xb1, 0x5b, 0x6a, 0xcb, 0xbe, 0x39, 0x4a, 0x4c, 0x58, 0xcf,
		0xd0, 0xef, 0xaa, 0xfb, 0x43, 0x4d, 0x33, 0x85, 0x45, 0xf9, 0x02, 0x7f, 0x50, 0x3c, 0x9f, 0xa8,
		0x51, 0xa3, 0x40, 0x8f, 0x92, 0x9d, 0x38, 0xf5, 0xbc, 0xb6, 0xda, 0x21, 0x10, 0xff, 0xf3, 0xd2,
		0xcd, 0x0c, 0x13, 0xec, 0x5f, 0x97, 0x44, 0x17, 0xc4, 0xa7, 0x7e, 0x3d, 0x64, 0x5d, 0x19, 0x73,
		0x60, 0x81, 0x4f, 0xdc, 0x22, 0x2a, 0x90, 0x88, 0x46, 0xee, 0xb8, 0x14, 0xde, 0x5e, 0x0b, 0xdb,
		0xe0, 0x32, 0x3a, 0x0a, 0x49, 0x06, 0x24, 0x5c, 0xc2, 0xd3, 0xac, 0x62, 0x91, 0x95, 0xe4, 0x79,
		0xe7, 0xc8, 0x37, 0x6d, 0x8d, 0xd5, 0x4e, 0xa9, 0x6c, 0x56, 0xf4, 0xea, 0x65, 0x7a, 0xae, 0x08,
		0xba, 0x78, 0x25, 0x2e, 0x1c, 0xa6, 0xb4, 0xc6, 0xe8, 0xdd, 0x74, 0x1f, 0x4b, 0xbd, 0x8b, 0x8a,
		0x70, 0x3e, 0xb5, 0x66, 0x48, 0x03, 0xf6, 0x0e, 0x61, 0x35, 0x57, 0xb9, 0x86, 0xc1, 0x1d, 0x9e,
		0xe1, 0xf8, 0x98, 0x11, 0x69, 0xd9, 0x8e, 0x94, 0x9b, 0x1e, 0x87, 0xe9, 0xce, 0x55, 0x28, 0xdf,
		0x8c, 0xa1, 0x89, 0x0d, 0xbf, 0xe6, 0x42, 0x68, 0x41, 0x99, 0x2d, 0x0f, 0xb0, 0x54, 0xbb, 0x16,
	};
	return __sbox[in];

}

__device__ uint8_t inverse_sbox(uint8_t in) {
	const static uint8_t __inverse_sbox[] = {
		0x52, 0x09, 0x6a, 0xd5, 0x30, 0x36, 0xa5, 0x38, 0xbf, 0x40, 0xa3, 0x9e, 0x81, 0xf3, 0xd7, 0xfb,
		0x7c, 0xe3, 0x39, 0x82, 0x9b, 0x2f, 0xff, 0x87, 0x34, 0x8e, 0x43, 0x44, 0xc4, 0xde, 0xe9, 0xcb,
		0x54, 0x7b, 0x94, 0x32, 0xa6, 0xc2, 0x23, 0x3d, 0xee, 0x4c, 0x95, 0x0b, 0x42, 0xfa, 0xc3, 0x4e,
		0x08, 0x2e, 0xa1, 0x66, 0x28, 0xd9, 0x24, 0xb2, 0x76, 0x5b, 0xa2, 0x49, 0x6d, 0x8b, 0xd1, 0x25,
		0x72, 0xf8, 0xf6, 0x64, 0x86, 0x68, 0x98, 0x16, 0xd4, 0xa4, 0x5c, 0xcc, 0x5d, 0x65, 0xb6, 0x92,
		0x6c, 0x70, 0x48, 0x50, 0xfd, 0xed, 0xb9, 0xda, 0x5e, 0x15, 0x46, 0x57, 0xa7, 0x8d, 0x9d, 0x84,
		0x90, 0xd8, 0xab, 0x00, 0x8c, 0xbc, 0xd3, 0x0a, 0xf7, 0xe4, 0x58, 0x05, 0xb8, 0xb3, 0x45, 0x06,
		0xd0, 0x2c, 0x1e, 0x8f, 0xca, 0x3f, 0x0f, 0x02, 0xc1, 0xaf, 0xbd, 0x03, 0x01, 0x13, 0x8a, 0x6b,
		0x3a, 0x91, 0x11, 0x41, 0x4f, 0x67, 0xdc, 0xea, 0x97, 0xf2, 0xcf, 0xce, 0xf0, 0xb4, 0xe6, 0x73,
		0x96, 0xac, 0x74, 0x22, 0xe7, 0xad, 0x35, 0x85, 0xe2, 0xf9, 0x37, 0xe8, 0x1c, 0x75, 0xdf, 0x6e,
		0x47, 0xf1, 0x1a, 0x71, 0x1d, 0x29, 0xc5, 0x89, 0x6f, 0xb7, 0x62, 0x0e, 0xaa, 0x18, 0xbe, 0x1b,
		0xfc, 0x56, 0x3e, 0x4b, 0xc6, 0xd2, 0x79, 0x20, 0x9a, 0xdb, 0xc0, 0xfe, 0x78, 0xcd, 0x5a, 0xf4,
		0x1f, 0xdd, 0xa8, 0x33, 0x88, 0x07, 0xc7, 0x31, 0xb1, 0x12, 0x10, 0x59, 0x27, 0x80, 0xec, 0x5f,
		0x60, 0x51, 0x7f, 0xa9, 0x19, 0xb5, 0x4a, 0x0d, 0x2d, 0xe5, 0x7a, 0x9f, 0x93, 0xc9, 0x9c, 0xef,
		0xa0, 0xe0, 0x3b, 0x4d, 0xae, 0x2a, 0xf5, 0xb0, 0xc8, 0xeb, 0xbb, 0x3c, 0x83, 0x53, 0x99, 0x61,
		0x17, 0x2b, 0x04, 0x7e, 0xba, 0x77, 0xd6, 0x26, 0xe1, 0x69, 0x14, 0x63, 0x55, 0x21, 0x0c, 0x7d,
	};
	return __inverse_sbox[in];

}

#endif

// AES core functions block
#if 1
__host__ uint8_t *key_schedule(const uint8_t *input_key) {
	uint8_t *output = (uint8_t *)malloc((10 + 1) * 16 * sizeof(uint8_t));
	uint8_t temp[4], *cursor_4_before, *cursor_16_before, *cursor_current;
	size_t counter, i, j;

	// Copy the first 16 uint8_ts
	memcpy(output, input_key, 16 * sizeof(uint8_t));

	// Initialize cursors.
	cursor_current = output + 16;
	cursor_4_before = cursor_current - 4;
	cursor_16_before = cursor_current - 16;

	// Rock and roll.

	for (counter = 0; counter != 10; counter++) {
		// RotWord
		temp[3] = cursor_4_before[0];
		temp[0] = cursor_4_before[1];
		temp[1] = cursor_4_before[2];
		temp[2] = cursor_4_before[3];

		// SubBytes and add
		for (i = 0; i != 4; ++i) {
			cursor_current[i] = sbox(temp[i]) ^ cursor_16_before[i];
			if (i == 0) {
				cursor_current[i] ^= round_const[counter];
			}
		}

		// Forward all cursors
		cursor_current += 4, cursor_4_before += 4, cursor_16_before += 4;

		// Three step remaining is simple.
		for (i = 0; i != 3; ++i) {
			for (j = 0; j != 4; ++j) {
				cursor_current[j] = cursor_4_before[j] ^ cursor_16_before[j];
			}
			cursor_current += 4, cursor_4_before += 4, cursor_16_before += 4;
		}
	}

	return output;
}

__device__ void sub_bytes(uint8_t *input) {
	int i = 15;
	do {
		input[i] = sbox(input[i]);
	} while (--i >= 0);
}

__device__ void inverse_sub_bytes(uint8_t *input) {
	int i = 15;
	do {
		input[i] = inverse_sbox(input[i]);
	} while (--i >= 0);
}

__device__ void shift_rows(uint8_t *input) {
	int index, inner_index;
#define OFFSET(i, j) ((j) * 4 + (i))
	for (index = 0; index != 4; ++index) {
		uint8_t temp;
		inner_index = index;
		while (inner_index-- > 0) {
			temp = input[OFFSET(index, 0)];
			input[OFFSET(index, 0)] = input[OFFSET(index, 1)];
			input[OFFSET(index, 1)] = input[OFFSET(index, 2)];
			input[OFFSET(index, 2)] = input[OFFSET(index, 3)];
			input[OFFSET(index, 3)] = temp;
		}
	}
#undef OFFSET
}

__device__ void inverse_shift_rows(uint8_t *input) {
	int index, inner_index;
#define OFFSET(i, j) ((j) * 4 + (i))
	for (index = 0; index != 4; ++index) {
		uint8_t temp;
		inner_index = index;
		while (inner_index-- > 0) {
			temp = input[OFFSET(index, 3)];
			input[OFFSET(index, 3)] = input[OFFSET(index, 2)];
			input[OFFSET(index, 2)] = input[OFFSET(index, 1)];
			input[OFFSET(index, 1)] = input[OFFSET(index, 0)];
			input[OFFSET(index, 0)] = temp;
		}
	}
#undef OFFSET
}

__device__ uint8_t mul2(uint8_t a) {
	static uint8_t cache[256] = {
		0x00, 0x02, 0x04, 0x06, 0x08, 0x0A, 0x0C, 0x0E, 0x10, 0x12, 0x14, 0x16, 0x18, 0x1A, 0x1C, 0x1E,
		0x20, 0x22, 0x24, 0x26, 0x28, 0x2A, 0x2C, 0x2E, 0x30, 0x32, 0x34, 0x36, 0x38, 0x3A, 0x3C, 0x3E,
		0x40, 0x42, 0x44, 0x46, 0x48, 0x4A, 0x4C, 0x4E, 0x50, 0x52, 0x54, 0x56, 0x58, 0x5A, 0x5C, 0x5E,
		0x60, 0x62, 0x64, 0x66, 0x68, 0x6A, 0x6C, 0x6E, 0x70, 0x72, 0x74, 0x76, 0x78, 0x7A, 0x7C, 0x7E,
		0x80, 0x82, 0x84, 0x86, 0x88, 0x8A, 0x8C, 0x8E, 0x90, 0x92, 0x94, 0x96, 0x98, 0x9A, 0x9C, 0x9E,
		0xA0, 0xA2, 0xA4, 0xA6, 0xA8, 0xAA, 0xAC, 0xAE, 0xB0, 0xB2, 0xB4, 0xB6, 0xB8, 0xBA, 0xBC, 0xBE,
		0xC0, 0xC2, 0xC4, 0xC6, 0xC8, 0xCA, 0xCC, 0xCE, 0xD0, 0xD2, 0xD4, 0xD6, 0xD8, 0xDA, 0xDC, 0xDE,
		0xE0, 0xE2, 0xE4, 0xE6, 0xE8, 0xEA, 0xEC, 0xEE, 0xF0, 0xF2, 0xF4, 0xF6, 0xF8, 0xFA, 0xFC, 0xFE,
		0x1B, 0x19, 0x1F, 0x1D, 0x13, 0x11, 0x17, 0x15, 0x0B, 0x09, 0x0F, 0x0D, 0x03, 0x01, 0x07, 0x05,
		0x3B, 0x39, 0x3F, 0x3D, 0x33, 0x31, 0x37, 0x35, 0x2B, 0x29, 0x2F, 0x2D, 0x23, 0x21, 0x27, 0x25,
		0x5B, 0x59, 0x5F, 0x5D, 0x53, 0x51, 0x57, 0x55, 0x4B, 0x49, 0x4F, 0x4D, 0x43, 0x41, 0x47, 0x45,
		0x7B, 0x79, 0x7F, 0x7D, 0x73, 0x71, 0x77, 0x75, 0x6B, 0x69, 0x6F, 0x6D, 0x63, 0x61, 0x67, 0x65,
		0x9B, 0x99, 0x9F, 0x9D, 0x93, 0x91, 0x97, 0x95, 0x8B, 0x89, 0x8F, 0x8D, 0x83, 0x81, 0x87, 0x85,
		0xBB, 0xB9, 0xBF, 0xBD, 0xB3, 0xB1, 0xB7, 0xB5, 0xAB, 0xA9, 0xAF, 0xAD, 0xA3, 0xA1, 0xA7, 0xA5,
		0xDB, 0xD9, 0xDF, 0xDD, 0xD3, 0xD1, 0xD7, 0xD5, 0xCB, 0xC9, 0xCF, 0xCD, 0xC3, 0xC1, 0xC7, 0xC5,
		0xFB, 0xF9, 0xFF, 0xFD, 0xF3, 0xF1, 0xF7, 0xF5, 0xEB, 0xE9, 0xEF, 0xED, 0xE3, 0xE1, 0xE7, 0xE5,
	};
	return cache[a];
}

__device__ void mix_columns(uint8_t *input) {
	int8_t i = 0;
	uint8_t temp[4], tmp;

	memcpy(temp, input, 4);

	for (i = 0; i < 16; i += 4) {
		temp[0] = input[i + 0];
		temp[1] = input[i + 1];
		temp[2] = input[i + 2];
		temp[3] = input[i + 3];
		tmp = temp[0] ^ temp[1] ^ temp[2] ^ temp[3];
		input[i + 0] = mul2(temp[0] ^ temp[1]) ^ temp[0] ^ tmp;
		input[i + 1] = mul2(temp[1] ^ temp[2]) ^ temp[1] ^ tmp;
		input[i + 2] = mul2(temp[2] ^ temp[3]) ^ temp[2] ^ tmp;
		input[i + 3] = mul2(temp[3] ^ temp[0]) ^ temp[3] ^ tmp;
	}
}

__device__ void inverse_mix_columns(uint8_t *input) {
	size_t i;
	uint8_t temp[4], tmp;

	for (i = 0; i != 16; i += 4) {
		temp[0] = input[i + 0];
		temp[1] = input[i + 1];
		temp[2] = input[i + 2];
		temp[3] = input[i + 3];
		tmp = temp[0] ^ temp[1] ^ temp[2] ^ temp[3];
		input[i + 0] = mul2(temp[0] ^ temp[1]) ^ temp[0] ^ tmp;
		input[i + 1] = mul2(temp[1] ^ temp[2]) ^ temp[1] ^ tmp;
		input[i + 2] = mul2(temp[2] ^ temp[3]) ^ temp[2] ^ tmp;
		input[i + 3] = mul2(temp[3] ^ temp[0]) ^ temp[3] ^ tmp;

		uint8_t u, v;
		u = mul2(mul2(temp[0] ^ temp[2]));
		v = mul2(mul2(temp[1] ^ temp[3]));
		tmp = mul2(v ^ u);

		input[i + 0] ^= tmp ^ u;
		input[i + 2] ^= tmp ^ u;
		input[i + 1] ^= tmp ^ v;
		input[i + 3] ^= tmp ^ v;
	}
}

#endif

__global__ void aes_128_single_block(uint8_t *input, uint8_t *round_keys, uint8_t *output, size_t size, size_t offset, size_t blockSize) {
	size_t index = blockDim.x * blockIdx.x + threadIdx.x;
	if (offset * blockSize + index >= size) {
		return;
	}
	size_t i, j;
	uint8_t *result, *data, *cursor_round_key;

	// forward pointers
	data = input + (offset * blockSize + index) * 16;
	result = output + (offset * blockSize + index) * 16;

	// initialize cursors.
	cursor_round_key = round_keys;

	// Copy plain text.
	for (i = 0; i != 16; ++i) {
		result[i] = data[i] ^ cursor_round_key[i];
	}
	cursor_round_key += 0x10;

	// Run rounds excluding the last round.
	for (i = 0; i != 9; ++i) {
		sub_bytes(result);
		shift_rows(result);
		mix_columns(result);
		// Add round
		for (j = 0; j != 16; ++j) {
			result[j] ^= cursor_round_key[j];
		}
		cursor_round_key += 0x10;
	}

	// Now the last round.
	sub_bytes(result);
	shift_rows(result);
	for (i = 0; i != 16; ++i) {
		result[i] = result[i] ^ cursor_round_key[i];
	}
}

__global__ void aes_decrypt_single_block(uint8_t *input, uint8_t *round_keys, uint8_t *output, size_t size, size_t offset, size_t blockSize) {
	size_t index = blockDim.x * blockIdx.x + threadIdx.x;
	if (offset * blockSize + index >= size) {
		return;
	}
	size_t i, j;
	uint8_t *result, *cursor_round_key, *data;

	// forward pointers
	result = output + (offset * blockSize + index) * 16;
	data = input + (offset * blockSize + index) * 16;
	cursor_round_key = round_keys + 160;

	// The last round
	for (i = 0; i != 16; ++i) {
		result[i] = data[i] ^ cursor_round_key[i];
	}

	inverse_shift_rows(result);
	inverse_sub_bytes(result);
	cursor_round_key -= 16;

	// Run rounds excluding the last round.
	for (i = 0; i != 9; ++i) {
		for (j = 0; j != 16; ++j) {
			result[j] = result[j] ^ cursor_round_key[j];
		}
		inverse_mix_columns(result);
		inverse_shift_rows(result);
		inverse_sub_bytes(result);
		cursor_round_key -= 16;
	}

	// Now the last addRoundKey.
	for (i = 0; i != 16; ++i) {
		result[i] = result[i] ^ cursor_round_key[i];
	}
}


hipError_t aes_on_cuda(const uint8_t *input, const uint8_t *keys, uint8_t *output, const size_t size, bool isEncrypt) {
	uint8_t *round_keys = key_schedule(keys);
	uint8_t *gpu_input = nullptr;
	uint8_t *gpu_round_keys = nullptr;
	uint8_t *gpu_output = nullptr;
	hipError_t cudaStatus;

#define ASSERT(s) \
	if (cudaStatus != hipSuccess) { \
		fprintf(stderr, s"\n"); \
		goto Error; \
	}


	// Select which GPU to work with.
	cudaStatus = hipSetDevice(0);
	ASSERT("hipSetDevice");

	// Prepare GPU memory block
	cudaStatus = hipMalloc((void**)&gpu_input, size * sizeof(uint8_t));
	ASSERT("hipMalloc");
	cudaStatus = hipMalloc((void**)&gpu_round_keys, (11 * 16) * sizeof(uint8_t));
	ASSERT("hipMalloc");
	cudaStatus = hipMalloc((void**)&gpu_output, size * sizeof(uint8_t));
	ASSERT("hipMalloc");

	// Copy data from memory to GPU memory.
	cudaStatus = hipMemcpy(gpu_input, input, size * sizeof(uint8_t), hipMemcpyHostToDevice);
	ASSERT("hipMemcpy: gpu_input");
	cudaStatus = hipMemcpy(gpu_round_keys, round_keys, (11 * 16) * sizeof(uint8_t), hipMemcpyHostToDevice);
	ASSERT("hipMemcpy: gpu_round_keys");

	size_t offset = 0;
	const size_t threadsPerBlock = 512;
	const size_t blockSize = 16;
	while (offset * threadsPerBlock * blockSize * 16 < size) {
		if (offset % 100 == 0) {
			printf("progress: %lf\r", offset * (double)threadsPerBlock * blockSize * 1600 / size);
		}
		if (isEncrypt) {
			aes_128_single_block <<<blockSize, threadsPerBlock >>>(gpu_input, gpu_round_keys, gpu_output, size / 16, offset, threadsPerBlock * blockSize);
		}
		else {
			aes_decrypt_single_block <<<blockSize, threadsPerBlock >>>(gpu_input, gpu_round_keys, gpu_output, size / 16, offset, threadsPerBlock * blockSize);
		}
		offset++;
	}

	cudaStatus = hipGetLastError();
	ASSERT("hipGetLastError");
	cudaStatus = hipDeviceSynchronize();
	ASSERT("hipDeviceSynchronize");
	cudaStatus = hipMemcpy(output, gpu_output, size * sizeof(uint8_t), hipMemcpyDeviceToHost);
	ASSERT("hipMemcpy");

Error:
	hipFree(gpu_input);
	hipFree(gpu_round_keys);
	hipFree(gpu_output);
	free(round_keys);

	return cudaStatus;

#undef ASSERT
}